#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <cstdlib>
#include <assert.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
 int i = blockIdx.x*blockDim.x + threadIdx.x;
 if (i < n) y[i] = a*x[i] + y[i];
}

__host__
void saxpy_cpu(int n, float a, float *x, float *y)
{
    for (int i=0; i<n; i++) {
        if (i < n) y[i] = a*x[i] + y[i];
    } 
}

int main(int argc, char** argv) {

    if (argc != 2 )
    {
        printf("You need to provide array size\n");
        assert(false); // or return -1;
    }
    int ARRAY_SIZE = atoi(argv[1]);
    
    float *x, *y, *z, *d_x, *d_y; //z for CPU calcualtions
    x = (float*)malloc(ARRAY_SIZE*sizeof(float));
    y = (float*)malloc(ARRAY_SIZE*sizeof(float));
    z = (float*)malloc(ARRAY_SIZE*sizeof(float));
    float a = 2.3f;

    hipMalloc(&d_x, ARRAY_SIZE*sizeof(float));
    hipMalloc(&d_y, ARRAY_SIZE*sizeof(float));

    for (int i =0; i<ARRAY_SIZE; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
        z[i] = 2.0f;
    }

    hipMemcpy(d_x, x, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);

    /* Run saxpy on CPU */
    printf("Computing SAXPY on the CPU… ");
    saxpy_cpu(ARRAY_SIZE, a, x, z);
    printf("Done!\n");
    
    printf("Computing SAXPY on the GPU… ");
    saxpy<<<(ARRAY_SIZE+255)/256, 256>>>(ARRAY_SIZE, a, d_x, d_y);
    hipMemcpy(y, d_y, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    printf("Done!\n");

    printf("Comparing the output for each implementation… ");

    float maxError = 0.0f;
    for (int i=0; i<ARRAY_SIZE; i++){
        maxError = max(maxError, abs(y[i]-z[i]));
    }

    if (maxError==0.0f) {
        printf("Correct!!\n");
    } else {
        printf("Not correct!!\n"); 
    }
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    free(z);
}