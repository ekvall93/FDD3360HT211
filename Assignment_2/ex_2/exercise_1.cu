#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"

__global__ void helloCuda() {
    printf("Hello from cuda device");
}

int main() {
    helloCuda << <1,1> > ();
    printf("Hello from CPU world...\n");
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}